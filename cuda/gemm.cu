// nvcc -lcublas gemm.cu && ./a.out 0

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>


// code is from siboehm's blog, with annotation for understanding

// matrix M*K @ K*N = M*N
// matmul is C = A@B
// gemm is C = α*(A@B)+β*C, * for scalar multiplication

__global__ void matmul(int M, int K, int N, const float* A, const float* B, float* C){

    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    // if statement is needed for tile quantization
    if(x < M && y < N){
        float tmp = 0.0;
        for(int i = 0; i < K; i++){
            tmp += A[x*K + i] * B[i*N + y];

        }

        C[x*N + y] = tmp;

    }

}


/*
Matrixes are stored in row-majored
When looping, A is consecutive in memory, B is not
*/

void runCublasFP32(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C){

	hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F, 
				N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

}

// naive implementation
__global__ void gemm_naive(int M, int K, int N, float alpha, const float* A, const float* B, float beta, float* C){

    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < M && y < N){
        float tmp = 0.0;
        for(int i = 0; i < K; i++){
            // load from A
            // load from B
            tmp += A[x*K + i] * B[i*N + y];
        }

        // load from C
        // store to C
        C[x*N + y] = alpha*tmp + beta*C[x*N + y];

    }

}

/*
In the previous kernel, threads are indexed as [threadIdx.x, threadIdx.y]
Note that in a naive warp launch, threadIdx.x "changes" faster than threadIdx.y

In total, we access A[x*K + i], B[i*N + y], and C[x*N + y]
since threadIdx.x changes faster, essentially we will have a bunch of threads on different rows of A
This means every thread requires a different row of A, but the same column on B

However, in the coalesced kernel, we set it so that all the warps are in the same row of A
This way, they require the same row of A, and different columns of B
Since B is row-major, this is actually good as it means we can access different columns of B contiguously

Accessing the same value is a within-warp broadcast,
different threads accessing different values in a contiguous block is referred to as coalesced memory access
*/

// global memory coalescing
// this blocksize is size of a warp
template <const uint BLOCKSIZE>
__global__ void gemm_global_mem_coalescing(int M, int K, int N, float alpha, const float* A, const float* B, float beta, float* C){

    // this ensures that 
    const uint x = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const uint y = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if(x < M && y < N){
        float tmp = 0.0;
        for(int i = 0; i < K; i++){
            tmp += A[x*K + i] * B[i*N + y];
        }
        C[x*N + y] = alpha*tmp + beta*C[x*N + y];
    }

}




#define M 8192
#define K 8192
#define N 8192

int CEIL_DIV(int m, int n){
    return (((m) + (n)-1) / (n));
}

bool verify_matrix(float *ref, float *mat, int size){
	for (int i = 0; i < size; i++){
		if (std::fabs(ref[i] - mat[i]) > 1e-3){
			std::cout << "failed at " << i << std::endl;
			return false;
		}
	}

	return true;
}

void print_float(float* values) {
    printf("float [%.6f, %.6f, %.6f, %.6f, %.6f, %.6f, %.6f, %.6f]\n",
           values[0], values[1], values[2], values[3],
           values[4], values[5], values[6], values[7]);
}

int main(int argc, char** argv){

    int kernel_num = 0;
    if (argc == 2){
        kernel_num = std::stoi(argv[1]);
    }

    // when initializing multiple pointers, you have to do this disgusting *varname convention blegh
    float *hA = nullptr, *hB = nullptr, *hC = nullptr, *C_ref = nullptr;
    float *dA = nullptr, *dB = nullptr, *dC = nullptr;

    hA = (float*)malloc(sizeof(float) * M * K);
	hB = (float*)malloc(sizeof(float) * K * N);
	hC = (float*)malloc(sizeof(float) * M * N);
	C_ref = (float*)malloc(sizeof(float) * M * N);

    hipMalloc((void**) &dA, sizeof(float)*M*K);
	hipMalloc((void**) &dB, sizeof(float)*K*N);
	hipMalloc((void**) &dC, sizeof(float)*M*N);

    std::ifstream f("/tmp/torch_gemm", std::ios::binary);
	if (!f.is_open()) {
        std::cerr << "Error opening file!\n";
        return -1;
    }

    f.read(reinterpret_cast<char*>(hA), sizeof(float)*M*K);
	f.read(reinterpret_cast<char*>(hB), sizeof(float)*K*N);
	f.read(reinterpret_cast<char*>(hC), sizeof(float)*K*N);
	f.read(reinterpret_cast<char*>(C_ref), sizeof(float)*M*N);
	f.close();

    hipMemcpy(dA, hA, sizeof(float)*M*K, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(float)*K*N, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, sizeof(float)*M*N, hipMemcpyHostToDevice);

    float alpha = 1.0f;
    float beta = 1.0f;

    hipblasHandle_t handle;
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
    dim3 blockDim(32, 32);
    switch(kernel_num){
    case 0:
        std::cout << "running cublas" << std::endl;
        if (hipblasCreate(&handle)){
            std::cerr << "Create cublas handle error" << std::endl;
            exit(EXIT_FAILURE);
        }
        runCublasFP32(handle, M, N, K, alpha, dA, dB, beta, dC);
        break;
    case 1:
        std::cout << "running naive implementation" << std::endl;
        gemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, dA, dB, beta, dC);
        break;
    case 2:
        std::cout << "global coalescing" << std::endl;
        gemm_global_mem_coalescing<32><<<gridDim, blockDim>>>(M, N, K, alpha, dA, dB, beta, dC);
        break;
    }

    // get content out of device
    hipMemcpy(hC, dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    std::cout << "torch reference: \t";
    print_float(C_ref);
    std::cout << "output: \t\t";
    print_float(hC);

    free(hA);
	free(hB);
	free(hC);
	free(C_ref);
    hipFree(dB);
    hipFree(dA);
    hipFree(dC);
	hipblasDestroy(handle);

    return 0;
}
